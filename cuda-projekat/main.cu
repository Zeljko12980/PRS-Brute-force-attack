#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <hip/hip_runtime.h>
#include "handshake_data.h"
//#include "crypto_kernels.cu"

#define PASSWORD_LENGTH 5
#define THREADS_PER_BLOCK 256

__global__ void brute_force_mic(bool* found, char* result);
__global__ void test_known_password(bool* found, char* result);

int main() {
    char* d_result;
    bool* d_found;

    hipMalloc(&d_result, PASSWORD_LENGTH + 1);
    hipMalloc(&d_found, sizeof(bool));
    hipMemset(d_found, 0, sizeof(bool));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Pokretanje pravog brute-force MIC validacionog kernela
    test_known_password<<<1, 1>>>(d_found, d_result);

    //brute_force_mic<<<400, THREADS_PER_BLOCK>>>(d_found, d_result);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    bool h_found;
    char h_result[PASSWORD_LENGTH + 1] = {0};

    hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(h_result, d_result, PASSWORD_LENGTH, hipMemcpyDeviceToHost);

    std::cout << "\n========== STATISTIKA ==========" << std::endl;
    std::cout << "Status: " << (h_found ? "Lozinka pronađena" : "Nije pronađena") << std::endl;
    if (h_found)
        std::cout << "Lozinka: " << h_result << std::endl;
    std::cout << "Vrijeme:  " << milliseconds << " ms" << std::endl;
    std::cout << "================================\n" << std::endl;

    hipFree(d_result);
    hipFree(d_found);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}